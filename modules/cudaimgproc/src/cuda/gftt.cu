#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include <thrust/execution_policy.h>
namespace cv { namespace cuda { namespace device
{
    namespace gfft
    {   

        texture<float, hipTextureType2D, hipReadModeElementType> eigTex(0, hipFilterModePoint, hipAddressModeClamp);

        __device__ int g_counter = 0;

        template <class Mask> __global__ void findCorners(float threshold, const Mask mask, float2* corners, int max_count, int rows, int cols)
        {   

            const int j = blockIdx.x * blockDim.x + threadIdx.x;
            const int i = blockIdx.y * blockDim.y + threadIdx.y;

            if (i > 0 && i < rows - 1 && j > 0 && j < cols - 1 && mask(i, j))
            {
                float val = tex2D(eigTex, j, i);

                if (val > threshold)
                {
                    float maxVal = val;

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i - 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j    , i - 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i - 1), maxVal);

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i), maxVal);

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i + 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j    , i + 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i + 1), maxVal);

                    if (val == maxVal)
                    {
                        const int ind = ::atomicAdd(&g_counter, 1);

                        if (ind < max_count)
                            corners[ind] = make_float2(j, i);
                    }
                }
            }


        }

        int findCorners_gpu(PtrStepSzf eig, float threshold, PtrStepSzb mask, float2* corners, int max_count, hipStream_t stream)
        {


            void* counter_ptr;

#ifdef HIP_TODO
            cudaSafeCall( hipGetSymbolAddress((void **)&counter_ptr, (const void *)g_counter) );
#endif
            cudaSafeCall( hipMemsetAsync(counter_ptr, 0, sizeof(int), stream) );

            bindTexture(&eigTex, eig);

            dim3 block(16, 16);
            dim3 grid(divUp(eig.cols, block.x), divUp(eig.rows, block.y));

            if (mask.data)
                hipLaunchKernelGGL((findCorners), dim3(grid), dim3(block), 0, stream, threshold, SingleMask(mask), corners, max_count, eig.rows, eig.cols);
            else
                hipLaunchKernelGGL((findCorners), dim3(grid), dim3(block), 0, stream, threshold, WithOutMask(), corners, max_count, eig.rows, eig.cols);

            cudaSafeCall( hipGetLastError() );

            int count;
            cudaSafeCall( hipMemcpyAsync(&count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            if (stream)
                cudaSafeCall(hipStreamSynchronize(stream));
            else
                cudaSafeCall( hipDeviceSynchronize() );
            return std::min(count, max_count);


        }

        class EigGreater
        {
        public:
            __device__ __forceinline__ bool operator()(float2 a, float2 b) const
            {   

                return tex2D(eigTex, a.x, a.y) > tex2D(eigTex, b.x, b.y);


                
            }
        };


        void sortCorners_gpu(PtrStepSzf eig, float2* corners, int count, hipStream_t stream)
        {

            bindTexture(&eigTex, eig);

            thrust::device_ptr<float2> ptr(corners);
#if THRUST_VERSION >= 100802
            if (stream)
                thrust::sort(thrust::cuda::par(ThrustAllocator::getAllocator()).on(stream), ptr, ptr + count, EigGreater());
            else
                thrust::sort(thrust::cuda::par(ThrustAllocator::getAllocator()), ptr, ptr + count, EigGreater());
#else
            thrust::sort(ptr, ptr + count, EigGreater());
#endif

        }
    } // namespace optical_flow
}}}


#endif /* CUDA_DISABLER */
